#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <iostream>
#include <vector>
#include <string>
#include <fstream>
#include <bitset>

#define silent false 	//no stderr except actual errors
#define verbal true		//print every component of every pixel before and after applying classification
#define visual false	//print avg of each pixel components in grid with img sides
#define debug  false	//do printf in kernel

#define INDEX_ERROR 800

#define EPS 1e-11

#define CSC(call)  																											\
do {																														\
	hipError_t err = call;																									\
	if (err != hipSuccess) {																								\
		std::cerr << "ERROR in " << __FILE__ << ":" << __LINE__ << ". Message: " << hipGetErrorString(err) << "\n";		\
		exit(0);																											\
	}																														\
} while(0)

//вариант 6
//Нахождение ранга матрицы

__global__ void kernel_gaussian_step(double* elements, int n, int m, int start_row_index, int active_colomn, double* result) {
	//n - количество строк (элементов в столбце)
	//m - количество столбцов (элементов в строке)

	int cur_row = blockIdx.x;
	int cur_col = threadIdx.x; 
	int in_row_offset = blockDim.x; //если закончились нити блока, а строка не закончилась
	int other_row_offset = gridDim.x; //если закончились блоки, а строки еще есть

	while(cur_row < n) {
		while(cur_col < m) { 
			double coef;
			if (cur_row > start_row_index) { //этот if не делит варпы, так что все норм
				coef = - elements[active_colomn*n + cur_row] / elements[active_colomn*n + start_row_index]; 
			}
			else {
				coef = 0; //нужно, чтобы потоки вообще не запускались для таких случаев 
			}


			result[cur_col*n + cur_row] = elements[cur_col*n + cur_row] + coef*elements[cur_col*n + start_row_index];
			if (debug) printf("KERNEL: cur_row = %d cur_col = %d element = %lf coef = %lf\n",cur_row,cur_col,elements[cur_col*n + cur_row],coef);
			cur_col += in_row_offset;
		}
		cur_row += other_row_offset;
	}
}

bool close_to_zero(double val) {
	if(val < EPS && val > -EPS) {
		return true;
	}
	return false;
}

//ПОМЕНЯТЬ 															!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
int find_max_elm(const double* array,int array_size) {
	//находит максимальный элемент массива и возвращает его индекс
	//ДОЛЖЕН ИСПОЛЬЗОВАТЬ THRUST

	double max_elm = abs(array[0]);
	int max_elm_pos = 0;

	for (int i=1; i < array_size; ++i) {
		if(abs(array[i]) > max_elm) {
			max_elm = abs(array[i]);
			max_elm_pos = i;
		}
	}

	return max_elm_pos;
}

class matrix{
	int n;
	int m;
	double* array;

public:
	matrix(int n_, int m_, double* array_) {
		n = n_;
		m = m_;
		array = array_;
	}

	matrix(int n_, int m_){ //считывание матрицы с stdin
		n = n_; m = m_;
		//n - количество строк (элементов в столбце)
		//m - количество столбцов (элементов в строке)
		double* arr_all = (double*)malloc(sizeof(double)*m*n);

		for (int i = 0; i < n; ++i){ //проход по строкам
			for (int j = 0; j < m; ++j){ //проход по столбцам
				double elm = 0;
				std::cin >> elm;
				arr_all[j*n + i] = elm;
			}
		}

		array = arr_all;
	}

	void print() {
		for (int i = 0; i < m; ++i) {
			for (int j = 0; j < n; ++j) {
				std::cout << array[i*n + j] << " ";
			}
			std::cout << "\n---\n";
		}
	}

	void printf() {
		for (int j = 0; j < n; ++j) {
			for (int i = 0; i < m; ++i) {
				//std::cout.precision(10);
				if(array[i*n + j] >= 0) {
					std::printf(" ");
				}
				std::printf("%.2lf ",array[i*n + j]);
				//std::cout << array[i*n + j] << " ";
			}
			std::printf("\n");
		}
	}

	//																							!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!
	void swap_rows(int lhs,int rhs) {
		//меняет местами строки lhs и rhs
		//должен работать на мультипроцессоре
		if(lhs >= n || rhs >= n) {
			throw INDEX_ERROR;
		}

		for (int i=0; i<m; ++i) {
			double swp = array[i*n + lhs];
			array[i*n + lhs] = array[i*n +rhs];	
			array[i*n + rhs] = swp;		
		}
	}

	//1 - определить ведущий элемент в столбце i (thrust)
	//2 - переставить строки местами (О(m)) (параллельно, одномерной сеткой)
	//3 - вычислить коэфециенты для каждой строки (O(n)) ДЕЛАЕТСЯ ВНУТРИ ЯДРА
	//4 - записать коэфициенты в разд. память. 
	//		Каждый блок работает с одной из строк, ему нужен только один коэфициент
	//		Изначально класть коэфициенты в глобальную память, затем, каждый блок доставет свой и кладет в свою разделяемую
	//		У каждого блока должен быть поток-лидер, который помещает нужный элемент в разделяемую память, остальные потоки должны начать работу только после завершения перемещения
	//		варп потока-лидера будет работать неоптимально
	//5 - преобразовать строки (O(n*n)) (параллельно, двумерной сеткой)

	int rank() {
		int rank = -1;
		int active_element_idx = 0;

		for (int i = 0; i < n - 1; ++i) { //i - текущая строка.

			int max_elm_idx = find_max_elm(&array[active_element_idx*n + i],n - i) + i;

			if(verbal) std::cerr << "swp\n" << "\tcur index " << i << "\n\tindex with max elm " << max_elm_idx <<"\n";

			swap_rows(i,max_elm_idx);
						
			double* device_matrix;
			CSC(hipMalloc (&device_matrix, sizeof(double)*m*n));
			CSC(hipMemcpy (device_matrix, array, sizeof(double)*m*n, hipMemcpyHostToDevice));
			
			double* device_result;
			CSC(hipMalloc (&device_result, sizeof(double)*m*n));


			kernel_gaussian_step<<<512,512>>> (device_matrix, n, m, i, active_element_idx, device_result);


			CSC(hipMemcpy (array, device_result, sizeof(double)*m*n, hipMemcpyDeviceToHost));
			
			if(verbal) std::cerr << "after transformation\n";
			if(verbal) printf();

			hipFree(device_matrix);
			hipFree(device_result);

			//следующая строка - i + 1
			//надо определить главный элемент в ннй
			//он точно будет дальше, чем, главный элемент в строке i

			bool zero_col = true;
			for (int probe_idx = active_element_idx; probe_idx < m; ++probe_idx) {
				for (int idx_in_col = i; idx_in_col < n - 1; ++idx_in_col) { //проверка, есть ли ненулевые элементы в рассматриваемой колонке. Если есть, то она будет основной.
					if(verbal) std::cerr  << array[probe_idx*n + idx_in_col + 1] << " ";
					if(!close_to_zero(array[probe_idx*n + idx_in_col + 1])) {
						zero_col = false;
						break;
					}
				}
				if(verbal) std::cerr << "\n";

				if(zero_col) {
					continue;
				}
				else {
					active_element_idx = probe_idx;
					break;
				}
			}

			if (zero_col) { //все следующие столбцы имеют нулевую активную часть
				rank = i + 1;
				break;
			}



			if(verbal) std::cerr << "new active element: " << active_element_idx << "\n";

			//определение активной колонки для следующего шага - нужны будут сравнения с нулем
		}

		if(rank == -1) {
			rank = n;
		}
		
		return rank;

	}

};

int main() {
	
	try{ 
		int n,m;
		std::cin >> n >> m;

		matrix matr(n,m);

		if(!silent) {
			std::cerr << "--\n";
			matr.print();
			std::cerr << "\n";
			matr.printf();
			std::cerr << "\n";
		}

		int rank = matr.rank();

		if(!silent) {
			std::cerr << "-- RANK: " << rank << " --\n";
		}
		std::cout << rank << "\n";
	}
	catch(int err) {
		if (err == 101) {
			std::cerr << "error opening file\n";
		} else
		if (err == 105){
			std::cerr << "error new length\n";
		} else 
		if (err == 800) {
			std::cerr << "error index\n";
		} else{
			std::cerr << "unknown error detected\n";
		}
	}

	return 0;
}


